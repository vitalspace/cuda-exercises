#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void multiply(int *a, int *b, int *c)
{
    *c = *a * *b;
}

int main()
{
    int h_a = 5;
    int h_b = 5;
    int h_c;

    int *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    multiply<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d * %d = %d\n", h_a, h_b, h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
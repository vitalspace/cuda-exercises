#include <stdio.h>
#include <hip/hip_runtime.h>


// Pude que el resultado no sea lo que esperas, recuerda que en CUDA o en este bloque de codigo se esta trabajando en paralelo
// por lo que no veras los numeros y mensajes consecutivamente. 

__global__ void fizzBuzz(int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (idx <= n) {
        if (idx % 3 == 0 && idx % 5 == 0) {
            printf("FizzBuzz\n");
        } else if (idx % 3 == 0) {
            printf("Fizz\n");
        } else if (idx % 5 == 0) {
            printf("Buzz\n");
        } else {
            printf("%d\n", idx);
        }
    }
}

int main() {
    const int n = 100;

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    fizzBuzz<<<numBlocks, blockSize>>>(n);

    hipDeviceSynchronize();

    return 0;
}
